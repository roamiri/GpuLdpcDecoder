#include "hip/hip_runtime.h"

#include <iostream>
#include "CGPU_Decoder_MS_SIMD.h"
#include "transpose/GPU_Transpose_uint8.h"

static const size_t BLOCK_SIZE = 128; 

CGPU_Decoder_MS_SIMD::CGPU_Decoder_MS_SIMD(size_t _nb_frames, size_t n, size_t k, size_t m):
CGPUDecoder(_nb_frames, n, k, m)
{
	struct hipDeviceProp_t devProp;
  	hipGetDeviceProperties(&devProp, 0);

	struct hipFuncAttributes attr;    
	hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(LDPC_Sched_Stage_1_MS_SIMD)); 

  	int nMP      = devProp.multiProcessorCount; // Number of STREAM PROCESSOR
  	int nWarp    = attr.maxThreadsPerBlock/32;  // PACKET threads EXECUTABLES PARALLEL
  	int nThreads = nWarp * 32;					// NUMBER DE THREAD MAXI PAR SP
  	int nDOF     = nb_frames;
  	int nBperMP  = 65536 / (attr.numRegs); 	// Nr of blocks per MP
  	int minB     = min(nBperMP*nThreads,1024);
  	int nBlocks  = max(minB/nThreads * nMP, nDOF/nThreads);  

}


CGPU_Decoder_MS_SIMD::~CGPU_Decoder_MS_SIMD()
{
	std::cout << "Destroy " << __FUNCTION__ << std::endl;
}

void CGPU_Decoder_MS_SIMD::decode(float Intrinsic_fix[_N], int Rprime_fix[_N], int number_iteration, bool stream)
{
    hipError_t Status;
	hipStream_t streams;
    if(stream)
    {
		hipStreamCreate(&streams);
	}

    size_t nb_blocks = nb_frames / BLOCK_SIZE;
	if( nb_frames % BLOCK_SIZE != 0 )
	{
		printf("(%ld - %ld)  (%ld - %ld)\n", nb_frames, BLOCK_SIZE, nb_frames/BLOCK_SIZE, nb_frames%BLOCK_SIZE);
		exit( 0 );
	}

    Status = hipMemcpy/*Async*/(d_MSG_C_2_V, Intrinsic_fix, sz_nodes * sizeof(float), hipMemcpyHostToDevice);
    ERROR_CHECK(Status, __FILE__, __LINE__);
	{
		dim3 grid(1, nb_frames/32);
		dim3 threads(32, 32);
		if(stream)
			Interleaver_uint8<<<grid, threads,0,streams>>>((int*)d_MSG_C_2_V, (int*)device_V, _N, nb_frames);
		else
			Interleaver_uint8<<<grid, threads>>>((int*)d_MSG_C_2_V, (int*)device_V, _N, nb_frames);
	}

	{
		if(stream)
			LDPC_Sched_Stage_1_MS_SIMD<<<nb_blocks, BLOCK_SIZE,0, streams>>>((unsigned int*)device_V, (unsigned int*)d_MSG_C_2_V, d_transpose, number_iteration);
		else
			LDPC_Sched_Stage_1_MS_SIMD<<<nb_blocks, BLOCK_SIZE>>>((unsigned int*)device_V, (unsigned int*)d_MSG_C_2_V, d_transpose, number_iteration);
	}
	{
		dim3 grid(1, nb_frames/32);
		dim3 threads(32, 32);
		if(stream)
			InvInterleaver_uint8<<<grid, threads,0, streams>>>((int*)device_V, (int*)d_MSG_C_2_V, _N, nb_frames);
		else
			InvInterleaver_uint8<<<grid, threads>>>((int*)device_V, (int*)d_MSG_C_2_V, _N, nb_frames);
	}
	Status = hipMemcpy(Rprime_fix, d_MSG_C_2_V, sz_nodes * sizeof(float), hipMemcpyDeviceToHost);
	ERROR_CHECK(Status, __FILE__, __LINE__);
}
